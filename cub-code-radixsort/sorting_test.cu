//#include "../../cub-1.8.0/hipcub/hipcub.hpp"   // or equivalently <cub/device/device_histogram.cuh>
#include "hipcub/hipcub.hpp"
#include "helper.cu.h"
#include "kernel.cu.h"

// template<class Z>
int validateZ(u_int32_t* A, uint32_t sizeAB) {
    int wrongCounter = 0;
    for(uint32_t i = 1; i < sizeAB; i++) {

      if (A[i-1] > A[i]){
        // printf("INVALID RESULT for i:%d, (A[i-1]=%d > A[i]=%d)\n", i, A[i-1], A[i]);
        wrongCounter++;
      }
    }
    return wrongCounter;
}

void randomInitNat(uint32_t* data, const uint32_t size, const uint32_t H) {
    for (int i = 0; i < size; ++i) {
        uint32_t r = rand();
        data[i] = r;
    }
}

double sortRedByKeyCUB( uint32_t* data_keys_in
                      , uint32_t* data_keys_out
                      , const uint64_t N
) {
    int beg_bit = 0;
    int end_bit = 32;

    void * tmp_sort_mem = NULL;
    size_t tmp_sort_len = 0;

    { // sort prelude
        hipcub::DeviceRadixSort::SortKeys( tmp_sort_mem, tmp_sort_len
                                      , data_keys_in, data_keys_out
                                      , N,   beg_bit,  end_bit
                                      );
        hipDeviceSynchronize();
        hipMalloc(&tmp_sort_mem, tmp_sort_len);
    }
    cudaCheckError();

    { // one dry run
        hipcub::DeviceRadixSort::SortKeys( tmp_sort_mem, tmp_sort_len
                                      , data_keys_in, data_keys_out
                                      , N,   beg_bit,  end_bit
                                      );
        hipDeviceSynchronize();
    }
    cudaCheckError();

    // timing
    double elapsed;
    struct timeval t_start, t_end, t_diff;
    gettimeofday(&t_start, NULL);

    for(int k=0; k<GPU_RUNS; k++) {
        hipcub::DeviceRadixSort::SortKeys( tmp_sort_mem, tmp_sort_len
                                      , data_keys_in, data_keys_out
                                      , N,   beg_bit,  end_bit
                                      );
    }
    hipDeviceSynchronize();
    cudaCheckError();

    gettimeofday(&t_end, NULL);
    timeval_subtract(&t_diff, &t_end, &t_start);
    elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / ((double)GPU_RUNS);

    hipFree(tmp_sort_mem);

    return elapsed;
}

// https://github.com/NVIDIA/cccl/blob/main/cub/cub/device/device_radix_sort.cuh
// Just copy-pasted the params from above (Not sure which ones we need for our implementation)
void radixSortKeys(
    void* d_temp_storage,
    size_t& temp_storage_bytes,
    u_int32_t* d_keys_in,
    u_int32_t* d_keys_out,
    u_int64_t num_items,
    int begin_bit,
    int end_bit
) {
    const int B = 256; // CUDA block size
    const int Q = 22; // elements processed by each thread
    const int lgH = 8; // bits sorted at a time
    const int H = 256; // Histogram size

    int numBlocks = (num_items + B * Q - 1) / (B * Q);
    // printf("numBlocks: %u\n", numBlocks);
    int threadsPerBlock = B;

    uint32_t *histogram_res = (uint32_t*) malloc(numBlocks * H * sizeof(uint32_t));
    uint32_t *histogram;
    cudaSucceeded(hipMalloc((void**) &histogram, numBlocks * H * sizeof(uint32_t)));
    uint32_t *transpose_res;
    cudaSucceeded(hipMalloc((void**) &transpose_res, numBlocks * H * sizeof(uint32_t)));
    uint32_t *final_transpose_res;
    cudaSucceeded(hipMalloc((void**) &final_transpose_res, numBlocks * H * sizeof(uint32_t)));

    hipMemcpy(d_keys_out, d_keys_in, num_items * sizeof(uint32_t), hipMemcpyDeviceToDevice);
    hipDeviceSynchronize();

    for (int i = 0; i < 4; i++) {
        // Step 1. (Also writes back to global memory)
        sortTile<Q,B,lgH><<<numBlocks, threadsPerBlock>>>(d_keys_out, i, num_items);
        hipDeviceSynchronize();

        // Step 2.
        histogramKernel<H, B><<<numBlocks, threadsPerBlock>>>(d_keys_out, histogram, Q, i, num_items);
        hipDeviceSynchronize();

        // Step 3. transpose -> scan -> transpose
        const int TILE = 16;
        int dimy = (numBlocks + TILE - 1) / TILE;
        int dimx = (H + TILE - 1) / TILE;
        dim3 block(TILE, TILE, 1), grid(dimx, dimy, 1);

        transposeKernel<TILE><<<grid, block>>>(histogram, transpose_res, H, numBlocks);
        hipDeviceSynchronize();
        scanKernel<<<numBlocks, threadsPerBlock>>>(transpose_res, H, numBlocks);
        hipDeviceSynchronize();
        transposeKernel<TILE><<<grid, block>>>(transpose_res, final_transpose_res, H, numBlocks);
        hipDeviceSynchronize();

        // Step 4. Scatter sorted tiles to global positions
        scatter<Q,B,lgH><<<numBlocks, threadsPerBlock>>>(d_keys_out, histogram, final_transpose_res, i, num_items);
        hipDeviceSynchronize();
    }

    hipFree(histogram);
}



double radixSortBench( uint32_t* data_keys_in
                      , uint32_t* data_keys_out
                      , const uint64_t N
) {
    int beg_bit = 0;
    int end_bit = 32;

    void * tmp_sort_mem = NULL;
    size_t tmp_sort_len = 0;


    { // sort prelude
        radixSortKeys( tmp_sort_mem, tmp_sort_len
                                      , data_keys_in, data_keys_out
                                      , N,   beg_bit,  end_bit
                                      );
        hipMalloc(&tmp_sort_mem, tmp_sort_len);
    }
    cudaCheckError();

    { // one dry run
        radixSortKeys( tmp_sort_mem, tmp_sort_len
                                      , data_keys_in, data_keys_out
                                      , N,   beg_bit,  end_bit
                                      );
        hipDeviceSynchronize();
    }
    cudaCheckError();


    // timing
    double elapsed;
    struct timeval t_start, t_end, t_diff;
    gettimeofday(&t_start, NULL);

    for(int k=0; k<GPU_RUNS; k++) {
        radixSortKeys( tmp_sort_mem, tmp_sort_len
                                      , data_keys_in, data_keys_out
                                      , N,   beg_bit,  end_bit
                                      );
    }
    hipDeviceSynchronize();
    cudaCheckError();

    gettimeofday(&t_end, NULL);
    timeval_subtract(&t_diff, &t_end, &t_start);
    elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / ((double)GPU_RUNS);

    hipFree(tmp_sort_mem);

    return elapsed;
}

int main (int argc, char * argv[]) {
    if (argc != 3) {
        printf("Usage: %s <size-of-array> <baseline>(1 for baseline)\n", argv[0]);
        exit(1);
    }

    const uint32_t N = atoi(argv[1]);
    const uint64_t BASELINE = atoi(argv[2]);

    //Allocate and Initialize Host data with random values
    uint32_t *h_keys = (uint32_t*) malloc(N*sizeof(uint32_t));//{2, 3, 50, 1, 10, 5, 667, 3, 78, 23, 100};
    uint32_t* h_keys_res  = (uint32_t*) malloc(N*sizeof(uint32_t));
    randomInitNat(h_keys, N, N/10);

    //Allocate and Initialize Device data
    uint32_t* d_keys_in;
    uint32_t* d_keys_out;
    cudaSucceeded(hipMalloc((void**) &d_keys_in,  N * sizeof(uint32_t)));
    cudaSucceeded(hipMemcpy(d_keys_in, h_keys, N * sizeof(uint32_t), hipMemcpyHostToDevice));
    cudaSucceeded(hipMalloc((void**) &d_keys_out, N * sizeof(uint32_t)));

    double elapsed;
    if(BASELINE) {
        elapsed = sortRedByKeyCUB( d_keys_in, d_keys_out, N );
    } else {
        elapsed = radixSortBench( d_keys_in, d_keys_out, N );
    }

    hipMemcpy(h_keys_res, d_keys_out, N*sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    cudaCheckError();

    int wrongCounter = validateZ(h_keys_res, N);
    printf("wrongCounter: %u\n", wrongCounter);
    bool success = wrongCounter == 0;

    if(BASELINE) {
        printf("Baseline (CUB) Sorting for N=%lu runs in: %.2f us, VALID: %d\n", N, elapsed, success);
    } else {
        printf("Radix Sorting for N=%lu runs in: %.2f us, VALID: %d\n", N, elapsed, success);
    }


    // Cleanup and closing
    hipFree(d_keys_in); hipFree(d_keys_out);
    //free(h_keys);
    free(h_keys_res);

    return success ? 0 : 1;
}
